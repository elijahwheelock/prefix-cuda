
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <unistd.h>
#include <errno.h>
#define log2(n) log(n)/log(2)

__global__ void short_prefix_sum(int *array, unsigned length, unsigned step) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ((i < length) && (i >= step)) {
        array[i] += array[i - step];
    }
    __syncthreads();
}

void short_prefix(int *host_array, unsigned length) {
    int array_size = length * sizeof(int);
    int *device_array; hipMalloc((void **) &device_array, array_size);
    hipMemcpy(device_array, host_array, array_size, hipMemcpyHostToDevice);
    
    dim3 numBlocks( length );
    dim3 threadsPerBlock(1);
    for (unsigned step=1; step<length; step<<=1) {
        short_prefix_sum<<<numBlocks, threadsPerBlock>>>(device_array, length, step);
        hipDeviceSynchronize();
    }
    
    hipMemcpy(host_array, device_array, array_size, hipMemcpyDeviceToHost);
    hipFree(device_array);
}

__global__ void long_prefix_upsweep(int *array, unsigned length, unsigned d) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int p = 1 << (d+1);
    if ((i < length) && (0 == (i % p))) {
        array[i + p - 1] += array[i + p/2 - 1];
    }
    __syncthreads();
}

__global__ void long_prefix_downsweep(int *array, unsigned length, unsigned d) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int p = 1 << (d+1);
    if ((i < length) && (0 == (i % p))) {
        int tmp = array[i + p/2 - 1];
        array[i + p/2 - 1] = array[i + p - 1];
        array[i + p - 1] = tmp + array[i + p/2 - 1];
    }
    __syncthreads();
}

void long_prefix(int *host_array, unsigned length) {
    int array_size = length * sizeof(int);
    int *device_array; hipMalloc((void **) &device_array, array_size);
    hipMemcpy(device_array, host_array, array_size, hipMemcpyHostToDevice);
    
    dim3 numBlocks(length / 1024);
    dim3 threadsPerBlock(1024);
    int  l = log2(length);
    for (int d=0; d < l; d++) {
        long_prefix_upsweep<<<numBlocks, threadsPerBlock>>>(device_array, length, d);
        hipDeviceSynchronize();
    }
    for (int d=l; d >= 0; d--) {
        long_prefix_downsweep<<<numBlocks, threadsPerBlock>>>(device_array, length, d);
        hipDeviceSynchronize();
    }
    
    hipMemcpy(host_array, device_array, array_size, hipMemcpyDeviceToHost);
    hipFree(device_array);
}

int main(int argc, char **argv){
    int input_n;
    if (2 == argc) {
        errno = 0;
        input_n = strtol(argv[1], NULL, 10);
    }
    else {
        fprintf(stderr, "argument must be exactly one integer\n");
        return 22;
    }
    if (errno) {
        fprintf(stderr, "error %d: %s\n", errno, strerror(errno));
        return errno;
    }
    int length = 1 << input_n;
    printf("length: %d\n", length);
    
    int array_size = length * sizeof(int);
    int *host_array = (int*) malloc(array_size);
    for (int i=0; i<length; ++i) {
        host_array[i] = 1;
    }
    
    long_prefix(host_array, length);
    
    bool not_expected = false;
    for (int i=0; i<length; ++i) {
        if (i != host_array[i]) {
            not_expected = true;
            break;
        }
    }
    if (not_expected) {
        for (int i=0; i<length; ++i) {
            printf("%d ", host_array[i]);
        }
        printf("failure!\n");
    } else { printf("success!\n"); }
    
    free(host_array);
}

